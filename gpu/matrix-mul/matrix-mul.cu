#include "hip/hip_runtime.h"
#include "readfile.h"
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
#define N 10000
// #define CHECK
#define RAND 200
#define INITIAL_SEED 12
void generate_matrix_data(){
    unsigned long long size = (unsigned long long)N*N*sizeof(double);
    double *a = (double*)malloc(size);
    double *b = (double*)malloc(size);
    srand(INITIAL_SEED);
    for( int row = 0; row < N; ++row ){
        for( int col = 0; col < N; ++col ){
            a[row*N + col] = (double)(rand() % RAND);
            b[row*N + col] = (double)(rand() % RAND);
        }
    }
    write_values_to_file("matrix_a_data",a,size);
    write_values_to_file("matrix_b_data",b,size);
    printf("generate data success\n");
}
__global__ void matrixMulGlobalKernel(double * pfMatrixA, double * pfMatrixB, double * pfMatrixC, int w)
{
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    double fCVal = 0.0f;
    for(int i =0; i < w; i++)
    {
        fCVal += pfMatrixA[nRow * w + i] * pfMatrixB[i * w + nCol];
    }
    pfMatrixC[nRow * w + nCol] = fCVal;
}
void matrixMulCPU(double * A, double * B, double* C, int w) {
    for (int i = 0; i < w; ++i) {
        for (int j = 0; j < w; ++j) {
            double sum = 0.0f;
            for (int k = 0; k < w; ++k) {
                sum += A[i * w + k] * B[k * w + j];
            }
            C[i * w + j] = sum;
        }
    }
}

int main(){
    hipError_t cudaStatus;
    unsigned long long size = (unsigned long long)N * N * sizeof (float );
    // Allocate input vectors h_A and h_B in host memory
    double* h_A = (double*)malloc(N * N * sizeof(double));
    double* h_B = (double*)malloc(N * N * sizeof(double));
    double* h_C = (double*)malloc(N * N * sizeof(double));
    double* h_C_cpu = (double*)malloc(N * N * sizeof(double));
    // Initialize input vectors
    read_values_from_file("matrix_a_data",h_A,size);
    read_values_from_file("matrix_b_data",h_B,size);
    // Allocate vectors in device memory

    double* d_A, * d_B, * d_C;
    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_B, N * N * sizeof(double));
    hipMalloc(&d_C, N * N * sizeof(double));

    hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(double), hipMemcpyHostToDevice);

    // Invoke kernel
    // 定义线程块和网格大小
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    // 调用 GPU 核函数，使用 double 类型
    double gpuStartTime = clock();
    matrixMulGlobalKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixAddGlobalKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return -1;
    }
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // Wait for the GPU to finish before proceeding
    hipDeviceSynchronize();
    double gpuEndTime = clock();
#ifdef CHECK
    double cpuStartTime = clock();
    matrixMulCPU( h_A, h_B, h_C_cpu,N);
    double cpuEndTime = clock();
    printf("GPU computation time: %lf\n", (gpuEndTime - gpuStartTime) / CLOCKS_PER_SEC);
    printf("CPU computation time: %lf\n", (cpuEndTime - cpuStartTime) / CLOCKS_PER_SEC);
    for (int i = 0; i < N * N; i++) {
        if (fabs(h_C_cpu[i] - h_C[i]) > 1e-10) {
            fprintf(stderr, "CPU and GPU results differ at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
#endif
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);
    return 0;
}