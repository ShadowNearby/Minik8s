#include "hip/hip_runtime.h"
#include "readfile.h"
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 32
#define N 10000
#define CHECK
#define RAND 200
#define INITIAL_SEED 12
void generate_matrix_data(){
    unsigned long long size = (unsigned long long)N*N*sizeof(double);
    double *a = (double*)malloc(size);
    double *b = (double*)malloc(size);
    srand(INITIAL_SEED);
    for( int row = 0; row < N; ++row ){
        for( int col = 0; col < N; ++col ){
            a[row*N + col] = (double)(rand() % RAND);
            b[row*N + col] = (double)(rand() % RAND);
        }
    }
    write_values_to_file("matrix_a_data",a,size);
    write_values_to_file("matrix_b_data",b,size);
    printf("generate data success\n");
}
__global__ void matrixAddGlobalKernel(double * pfMatrixA, double * pfMatrixB, double * pfMatrixC, int w)
{
    int nRow = blockIdx.y * blockDim.y + threadIdx.y;
    int nCol = blockIdx.x * blockDim.x + threadIdx.x;
    if (nRow < w && nCol < w)
        pfMatrixC[nRow * w + nCol] = pfMatrixA[nRow * w + nCol] + pfMatrixB[nRow * w + nCol];
}
void matrixAddCPU( double * a, double * b, double * c )
{

  for( int row = 0; row < N; ++row )
    for( int col = 0; col < N; ++col )
    {
      c[row * N + col] = a[row*N+col]+b[row*N+col];
    }
}
int main(){
    generate_matrix_data();
    hipError_t cudaStatus;
    unsigned long long size = (unsigned long long)N * N * sizeof (double );
    // Allocate input vectors h_A and h_B in host memory
    double * h_A = (double *)malloc(size);
    double * h_B = (double *)malloc(size);
    double * h_C = (double *)malloc(size);
    double * h_C_cpu = (double *)malloc(size);
    // Initialize input vectors
    read_values_from_file("matrix_a_data",h_A,size);
    read_values_from_file("matrix_b_data",h_B,size);
    // Allocate vectors in device memory
    double *d_A;
    hipMalloc(&d_A, size);
    double *d_B;
    hipMalloc(&d_B, size);
    double *d_C;
    hipMalloc(&d_C, size);
    double gpuStartTime = clock(); // 记录开始时间
    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (N + BLOCK_SIZE - 1) / BLOCK_SIZE);
    double gpuMemcpyTime = clock(); // 记录开始时间
    matrixAddGlobalKernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "matrixAddGlobalKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        return -1;
    }
    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    // Wait for the GPU to finish before proceeding
    hipDeviceSynchronize();
    double gpuEndTime = clock();
    #ifdef CHECK
    double cpuStartTime = clock();
    matrixAddCPU( h_A, h_B, h_C_cpu);
    double cpuEndTime = clock();
    printf("GPU copy memory time: %lf\n", (gpuMemcpyTime - gpuStartTime) / CLOCKS_PER_SEC);
    printf("GPU computation time: %lf\n", (gpuEndTime - gpuMemcpyTime) / CLOCKS_PER_SEC);
    printf("CPU computation time: %lf\n", (cpuEndTime - cpuStartTime) / CLOCKS_PER_SEC);
    for (int i = 0; i < N * N; i++) {
         if (fabs(h_C_cpu[i] - h_C[i]) > 1e-10) {
                fprintf(stderr, "CPU and GPU results differ at element %d!\n", i);
                exit(EXIT_FAILURE);
        }
    }
    #endif
    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    // Free host memory
    write_values_to_file("matrix_c_data",h_C,size);
    free(h_A);
    free(h_B);
    free(h_C);
    free(h_C_cpu);
}